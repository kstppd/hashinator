#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <chrono>
#include "../../src/hashinator/hashinator.h"
#define N  1<<18

typedef uint32_t val_type;

__global__
void fillMap(Hashinator<val_type,val_type> *dmap){
   int index = blockIdx.x * blockDim.x + threadIdx.x;
    dmap->set_element(index, index);
}

void cpuTest(){
   
   //timed block
   Hashinator<val_type,val_type> map;
   auto start = std::chrono::high_resolution_clock::now();
   for (val_type i=0; i<N;i++){
      map[i]=i;
   }
   map.print_all();
   auto end = std::chrono::high_resolution_clock::now();
   auto total_time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start);
   map.print_bank();
   printf("CPU time: %.3f seconds.\n", total_time.count() * 1e-9);
}

void gpuTest(int threads){
   //timed block
   Hashinator<val_type,val_type> map;
   map.resize(19);
   size_t total_keys=N;
   size_t total_threads=threads;
   size_t total_blocks= total_keys/total_threads;
   Hashinator<val_type,val_type>* dmap = map.upload();
   printf("Running with %i Threads and %i Blocks\n ",(int)threads,(int)total_blocks);
   auto start = std::chrono::high_resolution_clock::now();
   fillMap<<<total_blocks,total_threads>>>(dmap);
   hipDeviceSynchronize();
   auto end = std::chrono::high_resolution_clock::now();
   map.clean_up_after_device(dmap);
   map.print_all();
   auto total_time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start);
   printf("TIME: %.3f seconds for %zu elements at a load factor of %f\n", total_time.count() * 1e-9,map.size(),map.load_factor());


}

int main(){
   cpuTest();
   //gpuTest(32);
}

