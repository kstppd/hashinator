#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <chrono>
#include <random>
#include "../../include/hashinator/hashinator.h"
#include <gtest/gtest.h>
#include <random>
#include <algorithm>
#include <limits.h>

#define BLOCKSIZE 32
#define expect_true EXPECT_TRUE
#define expect_false EXPECT_FALSE
#define expect_eq EXPECT_EQ
constexpr int MINPOWER = 5;
constexpr int MAXPOWER = 20;


using namespace std::chrono;
using namespace Hashinator;
typedef uint32_t val_type;
typedef uint32_t key_type;
typedef split::SplitVector<hash_pair<key_type,val_type>,split::split_unified_allocator<hash_pair<val_type,val_type>>,split::split_unified_allocator<size_t>> vector ;
typedef split::SplitVector<key_type,split::split_unified_allocator<key_type>,split::split_unified_allocator<size_t>> ivector ;
typedef Hashmap<key_type,val_type> hashmap;


struct Predicate{
   HASHINATOR_HOSTDEVICE
   inline bool operator()( hash_pair<key_type,val_type>& element)const{
      return element.second%2==0;
   }
};

template <class Fn, class ... Args>
auto execute_and_time(const char* name,Fn fn, Args && ... args) ->bool{
   std::chrono::time_point<std::chrono::_V2::system_clock, std::chrono::_V2::system_clock::duration> start,stop;
   double total_time=0;
   start = std::chrono::high_resolution_clock::now();
   bool retval=fn(args...);
   stop = std::chrono::high_resolution_clock::now();
   auto duration = duration_cast<microseconds>(stop- start).count();
   total_time+=duration;
   //std::cout<<name<<" took "<<total_time<<" us"<<std::endl;
   return retval;
}


void create_input(vector& src, uint32_t bias=0){
   for (size_t i=0; i<src.size(); ++i){
      hash_pair<key_type,val_type>& kval=src.at(i);
      kval.first=i + bias;
      kval.second=i;
   }
}


void cpu_write(hashmap& hmap, vector& src){
   for (size_t i=0; i<src.size(); ++i){
      const hash_pair<key_type,val_type>& kval=src.at(i);
      hmap.at(kval.first)=kval.second;
   }
}

__global__ 
void gpu_write(hashmap* hmap, hash_pair<key_type,val_type>*src, size_t N){
   size_t index = blockIdx.x * blockDim.x + threadIdx.x;
   if (index < N ){
      hmap->set_element(src[index].first, src[index].second);
   }
}


__global__ 
void gpu_remove_insert(hashmap* hmap, hash_pair<key_type,val_type>*rm,  hash_pair<key_type,val_type>*add, size_t N){
   size_t index = blockIdx.x * blockDim.x + threadIdx.x;
   if (index  ==0 ){
      for ( int i =0; i <N ;++i ){
         hash_pair<key_type,val_type>elem=rm[i];
         auto rmval=hmap->read_element(elem.first);
         hmap->device_erase(elem.first);
      }
      for ( int i =0; i <N ;++i ){
         hash_pair<key_type,val_type>elem=add[i];
         hmap->set_element(elem.first,elem.second);
      }
   }
}


__global__
void gpu_delete_even(hashmap* hmap, hash_pair<key_type,val_type>*src,size_t N){
   size_t index = blockIdx.x * blockDim.x + threadIdx.x;
   if (index<N ){
      auto kpos=hmap->device_find(src[index].first);
      if (kpos==hmap->device_end()){assert(0 && "Catastrophic crash in deletion");}
      if (kpos->second %2==0 ){
         int retval=hmap->device_erase(kpos->first);
         assert(retval==1 && "Failed to erase!");
         retval=hmap->device_erase(kpos->first);
         assert(retval==0 && "Failed to not  erase!");

      }
   }
   return;
}

__global__
void gpu_recover_all_elements(hashmap* hmap,hash_pair<key_type,val_type>* src,size_t N  ){
   size_t index = blockIdx.x * blockDim.x + threadIdx.x;
   if (index < N ){
      key_type key= src[index].first;
      val_type val= src[index].second;
      auto it=hmap->device_find(key);
      if (it==hmap->device_end()){
         printf("END FOUND DEVICE\n");
         assert( 0 && "Failed in GPU RECOVER ALL ");
      }
      if (it->first!=key || it->second!=val){
         assert( 0 && "Failed in GPU RECOVER ALL ");
      }
   }
   return;
}


__global__
void gpu_recover_odd_elements(hashmap* hmap,hash_pair<key_type,val_type>* src,size_t N ){
   size_t index = blockIdx.x * blockDim.x + threadIdx.x;
   if (index < N ){
      key_type key= src[index].first;
      val_type val= src[index].second;
      if (val%2!=0){
         auto it=hmap->device_find(key);
         if (it==hmap->device_end()){
            assert( 0 && "Failed in GPU RECOVER ALL ");
         }
         if (it->first!=key || it->second!=val){
            assert( 0 && "Failed in GPU RECOVER ALL ");
         }
      }
   }

   //Iterate over all elements with 1 thread and check for evens;
   if (index==0){
      for (auto it=hmap->device_begin(); it!=hmap->device_end(); ++it){
         if (it->second%2==0 ){
            printf("Found even when there should not be any!\n");
            assert(0);
         }
      }
   }
   return;
}

bool recover_odd_elements(const hashmap& hmap, vector& src){
   for (size_t i=0; i<src.size(); ++i){
      const hash_pair<key_type,val_type>& kval=src.at(i);
      if (kval.second%2!=0){
         auto retval=hmap.find(kval.first);
         if (retval==hmap.end()){return false;}
         bool sane=retval->first==kval.first  &&  retval->second== kval.second ;
         if (!sane){ 
            return false; 
         }
      }
   }
   return true;
}

bool recover_all_elements(const hashmap& hmap, vector& src){
   for (size_t i=0; i<src.size(); ++i){
      const hash_pair<key_type,val_type>& kval=src.at(i);
      //std::cout<<"Validating "<<kval.first<<std::endl;
      auto retval=hmap.find(kval.first);
      if (retval==hmap.end()){
         std::cout<<"INVALID= "<<kval.first<<std::endl;
         return false;
      }
      bool sane=retval->first==kval.first  &&  retval->second== kval.second ;
      if (!sane){ 
         return false; 
      }
   //std::cout<<"Key validated "<<retval->first<<" "<<retval->second<<std::endl;
   }
   return true;
}

bool recover_odd_elements(hashmap* hmap, vector& src){
   for (size_t i=0; i<src.size(); ++i){
      const hash_pair<key_type,val_type>& kval=src.at(i);
      if (kval.second%2!=0){
         auto retval=hmap->find(kval.first);
         if (retval==hmap->end()){return false;}
         bool sane=retval->first==kval.first  &&  retval->second== kval.second ;
         if (!sane){ 
            return false; 
         }
      }
   }
   return true;
}

bool recover_all_elements(hashmap* hmap, vector& src){
   for (size_t i=0; i<src.size(); ++i){
      const hash_pair<key_type,val_type>& kval=src.at(i);
      auto retval=hmap->find(kval.first);
      if (retval==hmap->end()){return false;}
      bool sane=retval->first==kval.first  &&  retval->second== kval.second ;
      if (!sane){ 
         return false; 
      }
   }
   return true;
}

bool test_hashmap_1(int power){
   size_t N = 1<<power;
   size_t blocksize=BLOCKSIZE;
   size_t blocks=2*N/blocksize;

   bool cpuOK=true;

   //Create some input data
   vector src(N);
   create_input(src);
   hashmap hmap;
   hashmap* d_hmap;
   hmap.resize(power+1);

   //Upload to device and insert input
   d_hmap=hmap.upload();
   gpu_write<<<blocks,blocksize>>>(d_hmap,src.data(),src.size());
   hipDeviceSynchronize();
   hmap.download();

   //Verify all elements
   cpuOK=recover_all_elements(hmap,src);
   gpu_recover_all_elements<<<blocks,blocksize>>>(d_hmap,src.data(),src.size());
   hipDeviceSynchronize();
   return true;
   if (!cpuOK){
      return false;
   }

   //Delete some selection of the source data
   d_hmap=hmap.upload();
   gpu_delete_even<<<blocks,blocksize>>>(d_hmap,src.data(),src.size());
   hipDeviceSynchronize();
   hmap.download();

   //Quick check to verify there are no even elements
   for (const auto& kval : hmap){
      if (kval.second%2==0){
         std::cout<<kval.first<<" "<<kval.second<<std::endl;
         return false;
      }
   }
   
   //Verify odd elements;
   cpuOK=recover_odd_elements(hmap,src);
   gpu_recover_odd_elements<<<blocks,blocksize>>>(d_hmap,src.data(),src.size());
   hipDeviceSynchronize();
   if (!cpuOK){
      return false;
   }

   //Reinsert so that we can also test duplicate insertion
   d_hmap=hmap.upload();
   gpu_write<<<blocks,blocksize>>>(d_hmap,src.data(),src.size());
   hipDeviceSynchronize();
   //Download
   hmap.download();


   //Verify all elements
   cpuOK=recover_all_elements(hmap,src);
   gpu_recover_all_elements<<<blocks,blocksize>>>(d_hmap,src.data(),src.size());
   hipDeviceSynchronize();
   if (!cpuOK ){
      return false;
   }

   //If we made it to here we should be ok 
   return true;
}


bool test_hashmap_2(int power){
   size_t N = 1<<power;
   size_t blocksize=BLOCKSIZE;
   size_t blocks=2*N/blocksize;
   bool cpuOK=true;

   //Create some input data
   vector src(N);
   create_input(src);


   hashmap* hmap = new hashmap();
   hmap->resize(power+1);

   //Upload to device and insert input
   gpu_write<<<blocks,blocksize>>>(hmap,src.data(),src.size());
   hipDeviceSynchronize();

   //Verify all elements
   cpuOK=recover_all_elements(hmap,src);
   gpu_recover_all_elements<<<blocks,blocksize>>>(hmap,src.data(),src.size());
   hipDeviceSynchronize();
   if (!cpuOK ){
      return false;
   }

   //Delete some selection of the source data
   gpu_delete_even<<<blocks,blocksize>>>(hmap,src.data(),src.size());
   hipDeviceSynchronize();


   //Upload to device and insert input
   gpu_write<<<blocks,blocksize>>>(hmap,src.data(),src.size());
   hipDeviceSynchronize();

   //Upload to device and insert input
   gpu_write<<<blocks,blocksize>>>(hmap,src.data(),src.size());
   hipDeviceSynchronize();


   //Delete some selection of the source data
   gpu_delete_even<<<blocks,blocksize>>>(hmap,src.data(),src.size());
   hipDeviceSynchronize();

   //Quick check to verify there are no even elements
   for (const auto& kval : *hmap){
      if (kval.second%2==0){
         std::cout<<kval.first<<" "<<kval.second<<std::endl;
         return false;
      }
   }
   
   //Verify odd elements;
   cpuOK=recover_odd_elements(hmap,src);
   gpu_recover_odd_elements<<<blocks,blocksize>>>(hmap,src.data(),src.size());
   //hipDeviceSynchronize();
   if (!cpuOK){
      return false;
   }

   //Clean Tomstones and reinsert so that we can also test duplicate insertion
   hmap->clean_tombstones();
   gpu_write<<<blocks,blocksize>>>(hmap,src.data(),src.size());
   hipDeviceSynchronize();

   //Verify all elements
   cpuOK=recover_all_elements(hmap,src);
   gpu_recover_all_elements<<<blocks,blocksize>>>(hmap,src.data(),src.size());
   hipDeviceSynchronize();
   if (!cpuOK ){
      return false;
   }

   vector src2(N);
   create_input(src2);
   gpu_remove_insert<<<1,1>>>(hmap,src.data(),src2.data(),src.size());
   hipDeviceSynchronize();
   gpu_recover_all_elements<<<blocks,blocksize>>>(hmap,src2.data(),src2.size());
   hipDeviceSynchronize();

   delete hmap;
   hmap=nullptr;
   return true;
}

bool test_hashmap_3(int power){
   size_t N = 1<<power;

   //Create some input data
   vector src(N);
   create_input(src);
   hashmap hmap;
   bool cpuOK;

   for (auto i : src){
      hmap.insert(i);
   }

   cpuOK=recover_all_elements(hmap,src);
   if (!cpuOK){
      std::cout<<"Error at recovering all elements 1"<<std::endl;
      return false;
   }

   for (auto i:hmap){
      if (i.second%2==0){
         hmap.erase(i.first);
      }
   }

   cpuOK=recover_odd_elements(hmap,src);
   if (!cpuOK){
      std::cout<<"Error at recovering odd elements 2"<<std::endl;
      return false;
   }

   for (auto i : src){
      hmap.insert(i);
   }

   cpuOK=recover_all_elements(hmap,src);
   if (!cpuOK){
      std::cout<<"Error at recovering all elements 2"<<std::endl;
      return false;
   }
   return true;
}


bool test_hashmap_4(int power){
   size_t N = 1<<power;

   //Create some input data
   vector src(N);
   create_input(src);
   hashmap hmap;
   bool cpuOK;

   hmap.insert(src.data(),src.size());

   cpuOK=recover_all_elements(hmap,src);
   if (!cpuOK){
      std::cout<<"Error at recovering all elements 1"<<std::endl;
      return false;
   }

   //Get all even elements in src
   vector evenBuffer(src.size());
   ivector keyBuffer;
   split::tools::copy_if<hash_pair<key_type, val_type>,Predicate>(src,evenBuffer,Predicate());
   for (auto i:evenBuffer){
      keyBuffer.push_back(i.first);
   }


   //Erase using device
   hmap.erase(keyBuffer.data(),keyBuffer.size());

   cpuOK=recover_odd_elements(hmap,src);
   if (!cpuOK){
      std::cout<<"Error at recovering odd elements 2"<<std::endl;
      return false;
   }

   //Quick check to verify there are no even elements
   for (const auto& kval : hmap){
      if (kval.second%2==0){
         std::cout<<kval.first<<" "<<kval.second<<std::endl;
         return false;
      }
   }

   hipStream_t s ;
   hipStreamCreate(&s);
   hmap.clean_tombstones(s);
   cpuOK=recover_odd_elements(hmap,src);
   if (!cpuOK){
      std::cout<<"Error at recovering odd elements 2"<<std::endl;
      return false;
   }
   hmap.insert(src.data(),src.size());

   cpuOK=recover_all_elements(hmap,src);
   if (!cpuOK){
      std::cout<<"Error at recovering all elements 2"<<std::endl;
      return false;
   }
   return true;
}


TEST(HashmapUnitTets , Test1_HostDevice_UploadDownload){
   for (int power=MINPOWER; power<MAXPOWER; ++power){
      std::string name= "Power= "+std::to_string(power);
      bool retval = execute_and_time(name.c_str(),test_hashmap_1 ,power);
      expect_true(retval);
   }
}

TEST(HashmapUnitTets , Test2_HostDevice_New_Unified_Ptr){
   for (int power=MINPOWER; power<MAXPOWER; ++power){
      std::string name= "Power= "+std::to_string(power);
      bool retval = execute_and_time(name.c_str(),test_hashmap_2 ,power);
      expect_true(retval);
   }
}

TEST(HashmapUnitTets , Test3_Host){
   for (int power=MINPOWER; power<MAXPOWER; ++power){
      std::string name= "Power= "+std::to_string(power);
      bool retval = execute_and_time(name.c_str(),test_hashmap_3 ,power);
      expect_true(retval);
   }
}

TEST(HashmapUnitTets , Test4_DeviceKernels){
   for (int power=MINPOWER; power<MAXPOWER; ++power){
      std::string name= "Power= "+std::to_string(power);
      bool retval = execute_and_time(name.c_str(),test_hashmap_4 ,power);
      expect_true(retval);
   }
}

TEST(HashmapUnitTets ,Test_Clear_Perf_Host){

   const int sz=22;
   vector src(1<<sz);
   create_input(src);
   hashmap hmap(sz);
   bool cpuOK;
   hmap.insert(src.data(),src.size());
   cpuOK=recover_all_elements(hmap,src);
   if (!cpuOK){
      std::cout<<"Error at recovering all elements 1"<<std::endl;
      expect_true(false);
   }
   hmap.optimizeGPU();
   hipDeviceSynchronize();
   std::chrono::time_point<std::chrono::_V2::system_clock, std::chrono::_V2::system_clock::duration> start,stop;
   start = std::chrono::high_resolution_clock::now();
   hmap.clear();
   stop = std::chrono::high_resolution_clock::now();
   auto duration = duration_cast<microseconds>(stop- start).count();
   //std::cout<<"Clear took "<<duration<<" us status= "<<hmap.peek_status()<<std::endl;
}

TEST(HashmapUnitTets ,Test_Clear_Perf_Device){

   const int sz=22;
   vector src(1<<sz);
   create_input(src);
   hashmap hmap(sz);
   bool cpuOK;
   hmap.insert(src.data(),src.size());
   cpuOK=recover_all_elements(hmap,src);
   if (!cpuOK){
      std::cout<<"Error at recovering all elements 1"<<std::endl;
      expect_true(false);
   }
   hmap.optimizeGPU();
   hipDeviceSynchronize();
   std::chrono::time_point<std::chrono::_V2::system_clock, std::chrono::_V2::system_clock::duration> start,stop;
   start = std::chrono::high_resolution_clock::now();
   hmap.clear(targets::device);
   stop = std::chrono::high_resolution_clock::now();
   auto duration = duration_cast<microseconds>(stop- start).count();
   //std::cout<<"Clear took "<<duration<<" us status= "<<hmap.peek_status()<<std::endl;
}

TEST(HashmapUnitTets ,Test_Resize_Perf_Host){

   const int sz=24;
   vector src(1<<sz);
   create_input(src);
   hashmap hmap(sz);
   bool cpuOK;
   hmap.insert(src.data(),src.size());
   cpuOK=recover_all_elements(hmap,src);
   if (!cpuOK){
      std::cout<<"Error at recovering all elements 1"<<std::endl;
      expect_true(false);
   }
   hipDeviceSynchronize();
   std::chrono::time_point<std::chrono::_V2::system_clock, std::chrono::_V2::system_clock::duration> start,stop;
   start = std::chrono::high_resolution_clock::now();
   hmap.resize(sz+2);
   stop = std::chrono::high_resolution_clock::now();
   auto duration = duration_cast<microseconds>(stop- start).count();
   //std::cout<<"Resize took "<<duration<<" us status= "<<hmap.peek_status()<<std::endl;
}


TEST(HashmapUnitTets ,Test_Resize_Perf_Device){

   const int sz=24;
   vector src(1<<sz);
   create_input(src);
   hashmap hmap(sz);
   bool cpuOK;
   hmap.insert(src.data(),src.size());
   cpuOK=recover_all_elements(hmap,src);
   if (!cpuOK){
      std::cout<<"Error at recovering all elements 1"<<std::endl;
      expect_true(false);
   }
   hipDeviceSynchronize();
   std::chrono::time_point<std::chrono::_V2::system_clock, std::chrono::_V2::system_clock::duration> start,stop;
   start = std::chrono::high_resolution_clock::now();
   hmap.resize(sz+2,targets::device);
   stop = std::chrono::high_resolution_clock::now();
   auto duration = duration_cast<microseconds>(stop- start).count();
   //std::cout<<"Resize took "<<duration<<" us"<<std::endl;
   expect_true(hmap.peek_status()==status::success);
}


template <typename T, typename U>
struct Rule{
Rule(){}
   __host__ __device__
   inline bool operator()( hash_pair<T,U>& element)const{
      return element.first<1000;
   }
};


TEST(HashmapUnitTets ,Test_ErrorCodes_ExtractKeysByPattern){
   const int sz=5;
   vector src(1<<sz);
   create_input(src);
   hashmap hmap;
   hmap.insert(src.data(),src.size());
   bool cpuOK=recover_all_elements(hmap,src);
   expect_true(cpuOK);
   expect_true(hmap.peek_status()==status::success);
   ivector out;
   hmap.extractKeysByPattern(out,Rule<key_type,key_type>());
   for (auto i:out){
      expect_true(i<1000);
   }
}


TEST(HashmapUnitTets ,Test_Copy_Metadata){
   const int sz=18;
   vector src(1<<sz);
   create_input(src);
   hashmap hmap;
   hmap.insert(src.data(),src.size());
   bool cpuOK=recover_all_elements(hmap,src);
   expect_true(cpuOK);
   expect_true(hmap.peek_status()==status::success);
   Info* info;
   hipHostMalloc((void **) &info, sizeof(Info));
   hmap.copyMetadata(info);
   hipDeviceSynchronize();
   expect_true(1<<info->sizePower==hmap.bucket_count());
   expect_true(info->tombstoneCounter==hmap.tombstone_count());
   hipFree(info);

}

std::vector<key_type> generateUniqueRandomKeys(size_t size, size_t range=std::numeric_limits<int>::max()) {
    std::vector<key_type> elements;
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<int> dist(1, range);

    for (int i = 0; i < size; ++i) {
        key_type randomNum = i;//dist(gen);
        if (std::find(elements.begin(), elements.end(), randomNum) == elements.end()) {
            elements.push_back(randomNum);
        } else {
            --i;  
        }
    }
    return elements;
}

void insertDuplicates(std::vector<key_type>& vec, key_type element, size_t count) {
   if (count>0){
    vec.insert(vec.end(), count, element);
   }
   srand(time(NULL));
   std::random_shuffle(vec.begin(),vec.end());
}

TEST(HashmapUnitTets ,Test_Duplicate_Insertion){
   const int sz=10;
   for (size_t duplicates=2; duplicates<=(1<<sz);duplicates*=2){
      std::vector<key_type> keys=generateUniqueRandomKeys(1<<sz);

      for (size_t i = 0; i < duplicates;i++){
         insertDuplicates(keys,keys[0],1);
      }

      vector src(keys.size());
      for (size_t i =0;i<keys.size(); i++){
         src[i].first=keys[i];
         src[i].second=keys[i];
      }
      hashmap hmap;
      hmap.insert(src.data(),src.size(),1);
      bool cpuOK=recover_all_elements(hmap,src);
      expect_true(cpuOK);
      expect_true(hmap.peek_status()==status::success);
      expect_true(hmap.size()==((1<<sz)));
   }
}




int main(int argc, char* argv[]){
   srand(time(NULL));
   ::testing::InitGoogleTest(&argc, argv);
   return RUN_ALL_TESTS();
}
