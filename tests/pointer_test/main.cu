#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <chrono>
#include <gtest/gtest.h>
#include "../../include/splitvector/splitvec.h"
#include <hip/hip_runtime_api.h>
#include "../../include/splitvector/split_tools.h"
#define N 1024
#define expect_true EXPECT_TRUE
#define expect_false EXPECT_FALSE
#define expect_eq EXPECT_EQ
typedef split::SplitVector<int,split::split_unified_allocator<int>,split::split_unified_allocator<size_t>> vec ;

class Managed {
public:
   void *operator new(size_t len) {
      void *ptr;
      hipMallocManaged(&ptr, len);
      hipDeviceSynchronize();
      return ptr;
   }

   void operator delete(void *ptr) {
      hipDeviceSynchronize();
      hipFree(ptr);
   }

   void* operator new[] (size_t len) {
      void *ptr;
      hipMallocManaged(&ptr, len);
      hipDeviceSynchronize();
      return ptr;
   }

   void operator delete[] (void* ptr) {
      hipDeviceSynchronize();
      hipFree(ptr);
   }

};

class TestClass:public Managed{
   public:
   TestClass(){
      a= new vec(1024,128);
   }
   ~TestClass(){
      delete a;
   }
   vec* a;
};

__global__
void printClassVec(vec* a){
   printf("----> %d\n",(int)a->size());
   printf("----> %d\n",(int)a->at(12));
}

TEST(Test_GPU,VectorPrint){
   TestClass* test;
   test=new TestClass();
   printClassVec<<<1,1>>>(test->a);
   hipDeviceSynchronize();
   delete test;
}

int main(int argc, char* argv[]){
   ::testing::InitGoogleTest(&argc, argv);
   return RUN_ALL_TESTS();
}
