#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <chrono>
#include <unordered_set>
#include <random>
#include "../../include/hashinator/hashmap/hashmap.h"
constexpr int R = 10;

using namespace std::chrono;
using namespace Hashinator;
typedef uint32_t val_type;
typedef uint32_t key_type;
typedef split::SplitVector<hash_pair<key_type,val_type>> vector ;
typedef split::SplitVector<key_type> key_vec;
typedef split::SplitVector<val_type> val_vec;
using hashmap= Hashmap<key_type,val_type>;



auto generateNonDuplicatePairs(vector& src,const size_t size)->void {
    std::unordered_set<int> keys;
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<val_type> dist(1, std::numeric_limits<val_type>::max());

    src.clear();
    while (src.size() < size) {
        val_type key = dist(gen);
        // Check if the key is already present
        if (keys.find(key) == keys.end()) {
           val_type val=static_cast<val_type>(key/2);
            src.push_back({key,val});
            keys.insert(key);
        }
    }
}

auto generateNonDuplicatePairs(key_vec &keys,val_vec& vals,const size_t size)->void {
    std::unordered_set<int> unique_keys;
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<val_type> dist(1, std::numeric_limits<val_type>::max());
    keys.clear();
    vals.clear();
    while (keys.size() < size) {
        val_type key = dist(gen);
        // Check if the key is already present
        if (unique_keys.find(key) == unique_keys.end()) {
           val_type val=static_cast<val_type>(key/2);
            keys.push_back(key);
            vals.push_back(val);
            unique_keys.insert(key);
        }
    }
}

template <class Fn, class ... Args>
auto timeMe(Fn fn, Args && ... args){
   std::chrono::time_point<std::chrono::_V2::system_clock, std::chrono::_V2::system_clock::duration> start,stop;
   double total_time=0;
   start = std::chrono::high_resolution_clock::now();
   fn(args...);
   stop = std::chrono::high_resolution_clock::now();
   auto duration = duration_cast<milliseconds>(stop- start).count();
   total_time+=duration;
   return total_time;
}

void benchInsert(hashmap& hmap,key_type* gpuKeys, val_type* gpuVals,int sz){
   hmap.insert(gpuKeys,gpuVals,1<<sz,1);
   hmap.retrieve(gpuKeys,gpuVals,1<<sz);
   hmap.erase(gpuKeys,1<<sz);
   hmap.stats();
   hmap.clear();
   return ;
}

int main(int argc, char* argv[]){

   int sz= 24;
   if (argc>=2){
      sz=atoi(argv[1]);
   }
   hashmap hmap(sz+1);
   int device;
   split_gpuGetDevice(&device);
   hmap.memAdvise(hipMemAdviseSetPreferredLocation,device);
   hmap.memAdvise(hipMemAdviseSetAccessedBy,device);
   hmap.optimizeGPU();
   hmap.optimizeGPU();
   vector cpu_src;
   key_vec cpu_keys;
   val_vec cpu_vals;
   generateNonDuplicatePairs(cpu_keys,cpu_vals,1<<sz);
   std::cout<<"Generated "<<cpu_keys.size()<<" unique keys!"<<std::endl;

   key_type* gpuKeys;
   val_type* gpuVals;
   split_gpuMalloc((void **) &gpuKeys, (1<<sz)*sizeof(key_type));
   split_gpuMalloc((void **) &gpuVals, (1<<sz)*sizeof(val_type));
   split_gpuMemcpy(gpuKeys,cpu_keys.data(),(1<<sz)*sizeof(key_type),split_gpuMemcpyHostToDevice);
   split_gpuMemcpy(gpuVals,cpu_vals.data(),(1<<sz)*sizeof(key_type),split_gpuMemcpyHostToDevice);

   double t={0};
   for (int i =0; i<R; i++){
      hmap.optimizeGPU();
      t+=timeMe(benchInsert,hmap,gpuKeys,gpuVals,sz);
   }
   std::cout<<"Done in "<<t/R<<" ms"<<std::endl;

   split_gpuFree(gpuKeys);
   split_gpuFree(gpuVals);
   return 0;

}
